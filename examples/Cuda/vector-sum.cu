
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10
#define THREADS_PER_BLOCK 10

__global__ void gpuSum(int *a, int *b, int *c, int n) {
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  while (idx < n) {
    c[idx] = a[idx] + b[idx];
    idx += blockDim.x * gridDim.x;
  }
}

void fill_matrix(int *arr) {
  for (int i = 0 ; i < N ; i++) {
    for (int j = 0 ; j < N ; j++) {
      arr[(i * N) + j] = (i * N) + j;
    }
  }
}

void print_matrix(int *arr) {
  for (int i = 0 ; i < N ; i++) {
    for (int j = 0 ; j < N ; j++) {
      printf("%d\t", arr[(i * N) + j]);
    }
    printf("\n");
  }
  printf("\n");
}

int main() {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = sizeof(int) * N * N;

  // Allocate memory on host.
  a = (int*) malloc(size);
  b = (int*) malloc(size);
  c = (int*) malloc(size);

  // Allocate memory on device.
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
  hipMalloc((void**)&d_c, size);

  // Initialize array.
  fill_matrix(a);
  fill_matrix(b);

  // Copy from host to device.
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Perform sum on device.
  gpuSum<<<N*N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N*N);

  // Retrieve values from device to host.
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Print results.
  print_matrix(a);
  print_matrix(b);
  print_matrix(c);

  // Free host variables.
  free(a);
  free(b);
  free(c);

  // Free device variables.
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
