/*
*
*    Carlos Roman Rivera - A01700820
*
*    Programming Languages - Cuda Lab 1
*
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCKS 1000
#define THREADS_PER_BLOCK 512
#define RECTANGLES 1000000

__global__ void gpuPi(double *r, double width, int n) {
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);    // Index to calculate.
  int id = idx;                                         // My array position.
  double mid, height;                                   // Auxiliary variables.
  while (idx < n) {                                     // Dont overflow array.
    mid = (idx + 0.6) * width;                          // Formula.
    height = 4.0 / (1.0 + mid * mid);                   // Formula.
    r[id] += height;                                    // Store result.
    idx += (blockDim.x * gridDim.x);                    // Update index.
  }
}

int main() {
  double *pi;
  double *d_pi;
  double width;
  double result = 0;

  width = 1.0 / (double) RECTANGLES;

  int results = (BLOCKS * THREADS_PER_BLOCK);           // Total threads.

  int size = results * sizeof(double);                  // Size in bytes.

  pi = (double*) malloc(size);                          // Memory on host.

  hipMalloc((void**)&d_pi, size);                      // Memory on device.

  hipMemcpy(d_pi, pi, size, hipMemcpyHostToDevice);   // Host to device.

  gpuPi<<<BLOCKS/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_pi, width, RECTANGLES);

  hipMemcpy(pi, d_pi, size, hipMemcpyDeviceToHost);   // Device to host.

  for(int i = 0 ; i < results ; i++) {                  // Sum results.
    result += pi[i];
  }

  result *= width;                                      // Formula.

  printf("PI: %lf\n", result);                          // Display result.

  free(pi);                                             // Free host memory.

  hipFree(d_pi);                                       // Free device memory.

  return 0;
}
