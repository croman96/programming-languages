#include "hip/hip_runtime.h"
__global__ void dot(float *a, float *b, floar *c) {
  __shared__ float cache(THREADS_PER_BLOCK);

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int cacheIndex = threadIdx.x;

  float temp = 0 ;

  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = temp;

  __syncthreads();

  int i = blockDim.x / 2;

  while (i != 0) {
    if (cacheIndex < i) {
      cache[cacheIndex] += cache[cacheIndex + i];
    }
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0) {
    c[blockIdx.x] = cache[0];
  }
}
