/*
*
*    Carlos Roman Rivera - A01700820
*
*    Programming Languages - Cuda Quiz
*
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 9
#define K N/3
#define ThreadsPerBlock K
#define NumBlocks K

__global__ void compress(float *mat, int n, float *comp, int k){
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  if (row < k && col < k) {
    comp[col + row * k] = 0;
    for (int i_row = 0 ; i_row < k ; i_row++) {
      for (int j_col = 0 ; j_col < k ; j_col++) {
        comp[col + row * k] += mat[(col + j_col) + (row + i_row) * n];
      }
    }
  }

}

void print_mat(float *mat, int n){
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			printf("%.1f\t", mat[i*n+j]);
		}
		printf("\n");
	}
	printf("\n");
}

void fill_mat(float *mat, int n){
	int c = 0;
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			mat[i*n+j] = c++;
		}
	}
}

int main(){
	float *h_compress, *h_matrix;
	float *d_compress, *d_matrix;

	h_compress = (float *)malloc(sizeof(float) * K * K);
	h_matrix = (float *)malloc(sizeof(float) * N * N);

	fill_mat(h_matrix, N);
  fill_mat(h_compress, K);

	printf("Input matrix:\n");
	print_mat(h_matrix, N);

  hipMemcpy(d_matrix, h_matrix, sizeof(float) * N * N, hipMemcpyHostToDevice);
  hipMemcpy(d_compress, h_compress, sizeof(float) * K * K, hipMemcpyHostToDevice);

  dim3 Blocks(K,K);
	dim3 Threads(K,K);

  compress<<<Blocks, Threads>>>(d_matrix, N, d_compress, K);

  hipMemcpy(h_compress, d_compress, sizeof(float) * K * K, hipMemcpyDeviceToHost);

  printf("Compressed matrix:\n");
  print_mat(h_compress, K);

  free(h_matrix);
  free(h_compress);

  hipFree(d_matrix);
  hipFree(d_compress);

}
