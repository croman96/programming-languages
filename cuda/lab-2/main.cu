/*
*
*    Carlos Roman Rivera - A01700820
*
*    Programming Languages - Cuda Lab 2
*
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void matrix_multiplication(int *matrix_1, int *matrix_2, int *matrix_r, int m, int n, int p){

	int row = threadIdx.y + blockIdx.y * blockDim.y; 	// Multiply this row...
	int col = threadIdx.x + blockIdx.x * blockDim.x;	// with this column.

	// Matrix multiplication as follows:
	// (m x n) x (n x p) = (m x p)

	int id = row * p + col;	// Index of the result matrix in which we will write.
	int sum = 0;

	if (row < m && col < p) {
		for(int i = 0; i < n; i++) {
			// In matrix_1 we keep the row and advance in the columns.
			// In matrix_2 we keep the column and advance in the rows.
			sum = sum + matrix_1[row * n + i] * matrix_2[i * p + col];
			// row * n stays in the same row and  "+ i" advances 1 column each cicle.
			// i * p advances one row each cicle and  "+ col" keeps the same col.
		}
	 	matrix_r[id] = sum;
	}
}


// Display a matrix of the given dimensions.
void print_matrix(int *mat, int rows, int cols){
	for (int i = 0; i < rows; i++){
		for (int j = 0; j < cols; j++){
			printf("%d\t", mat[i * cols + j]);
		}
		printf("\n");
	}
	printf("\n");
}

// User gives the value of each element of the matrix.
void user_matrix(int *mat, int rows, int cols){
	int aux;
	for (int i = 0; i < rows; i++){
		for (int j = 0; j < cols; j++){
			printf("[%d][%d] = ", i, j);
			scanf("%i%*c", &aux);
			mat[i * cols + j] = aux;
		}
	}
}

// "Randomly" generate the value of each element of the matrix.
void fill_matrix(int *mat, int rows, int cols){
	for (int i = 0; i < rows; i++){
		for (int j = 0; j < cols; j++){
			mat[i * cols + j] = (rand() % 99) + 1;
		}
	}
}

int main(){

  srand(time(0));

	// Matrices
  int *h_matrix_1, *h_matrix_2, *h_matrix_r;
  int *d_matrix_1, *d_matrix_2, *d_matrix_r;

	// Dimensions
  int matrix_1_rows, matrix_1_cols;
  int matrix_2_rows, matrix_2_cols;

	// Memory size
  int matrix_1_size, matrix_2_size, matrix_r_size;

	// User input for whether randomly or manually initialize matrices.
	int respuesta;

  printf("Matrix 1 rows: ");
  scanf("%d%*c", &matrix_1_rows);

  printf("Matrix 1 cols: ");
  scanf("%d%*c", &matrix_1_cols);

  printf("\nMatrix 2 rows: ");
  scanf("%d%*c", &matrix_2_rows);

  printf("Matrix 2 cols: ");
  scanf("%d%*c", &matrix_2_cols);

	// Matrices must be (m x n) and (n x p)
	if (matrix_1_cols != matrix_2_rows) {
		printf("\nLas dimensiones introducidas no son aceptables.\n");
		return 0;
	}

	// Calculate memory given dimensions.
  matrix_1_size = sizeof(int) * matrix_1_rows * matrix_1_cols;
  matrix_2_size = sizeof(int) * matrix_2_rows * matrix_2_cols;
  matrix_r_size = sizeof(int) * matrix_1_rows * matrix_2_cols;

	// Allocate memory.
  h_matrix_1 = (int *)malloc(matrix_1_size);
  h_matrix_2 = (int *)malloc(matrix_2_size);
  h_matrix_r = (int *)malloc(matrix_r_size);


	// Select how to initialize matrices.
	printf("\nDeseas:\n1. Introducir matrices manualmente.\n2. Generar matrices aleatoriamente.\nR = ");

	scanf("%d%*c", &respuesta);

	if(respuesta == 1) {
		// User wants to initialize matrix.
		printf("\nMatriz A: \n");
		user_matrix(h_matrix_1, matrix_1_rows, matrix_1_cols);

		printf("\nMatriz B: \n");
		user_matrix(h_matrix_2, matrix_2_rows, matrix_2_cols);
	} else {
		// User wants random initialization.
		if (respuesta != 2) {
			// Invalid answer, therefore, randomly initialized.
			printf("\nOpcion invalida, generando aleatorias.\n");
		}
		fill_matrix(h_matrix_1, matrix_1_rows, matrix_1_cols);
	  fill_matrix(h_matrix_2, matrix_2_rows, matrix_2_cols);
	}

	// Display matrix for interactive purpose.
	printf("\nMatrix A:\n");
  print_matrix(h_matrix_1, matrix_1_rows, matrix_1_cols);

	// Display matrix for interactive purpose.
	printf("Matrix B:\n");
  print_matrix(h_matrix_2, matrix_2_rows, matrix_2_cols);

	// Allocate memory on device.
  hipMalloc((void**)&d_matrix_1, matrix_1_size);
  hipMalloc((void**)&d_matrix_2, matrix_2_size);
  hipMalloc((void**)&d_matrix_r, matrix_r_size);

	// Copy initialized matrices from host to device.
  hipMemcpy(d_matrix_1, h_matrix_1, matrix_1_size, hipMemcpyHostToDevice);
  hipMemcpy(d_matrix_2, h_matrix_2, matrix_2_size, hipMemcpyHostToDevice);

	// Each thread will calculate each element of the result matrix.
  int ThreadsPerBlock = matrix_2_cols;
  int NumBlocks = matrix_1_rows;

  dim3 Blocks(NumBlocks, NumBlocks);
  dim3 Threads(ThreadsPerBlock, ThreadsPerBlock);

	// Display for interactive purpose.
	printf("Blocks: %d\n", NumBlocks);
	printf("Threads/Block: %d\n", ThreadsPerBlock);

	// Execute on device.
  matrix_multiplication<<<Blocks, Threads>>>(d_matrix_1, d_matrix_2, d_matrix_r, matrix_1_rows, matrix_1_cols, matrix_2_cols);

	// Retrieve result from device and copy to host.
  hipMemcpy(h_matrix_r, d_matrix_r, matrix_r_size, hipMemcpyDeviceToHost);

	// Display results for illustrative purposes.
	printf("\n");
	printf("Matrix R:\n");
  print_matrix(h_matrix_r, matrix_1_rows, matrix_2_cols);

	// Free host memory.
  free(h_matrix_1);
  free(h_matrix_2);
  free(h_matrix_r);

	// Free device memory.
  hipFree(d_matrix_1);
  hipFree(d_matrix_2);
  hipFree(d_matrix_r);

}
